#include "MatrixUtil.cuh"

typedef std::complex<float>  cmplxF;
typedef std::complex<double> cmplxD;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

hipsolverStatus_t cusolverget_bufferSize(hipsolverHandle_t handle,
                      int m,
                      int n,
                      float *A,
                      int lda,
                      int *Lwork ) {
	return hipsolverDnSgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t cusolverget_bufferSize(hipsolverHandle_t handle,
                      int m,
                      int n,
                      double *A,
                      int lda,
                      int *Lwork ) {
	return hipsolverDnDgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t cusolverget_bufferSize(hipsolverHandle_t handle,
                      int m,
                      int n,
                      hipComplex *A,
                      int lda,
                      int *Lwork ) {
	return hipsolverDnCgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t cusolverget_bufferSize(hipsolverHandle_t handle,
                      int m,
                      int n,
                      hipDoubleComplex *A,
                      int lda,
                      int *Lwork ) {
	return hipsolverDnZgetrf_bufferSize(handle, m, n, A, lda, Lwork);
}

hipsolverStatus_t cusolvergetrf(hipsolverHandle_t handle,
           int m,
           int n,
           float *A,
           int lda,
           float *Workspace,
           int *devIpiv,
           int *devInfo) {
    return hipsolverDnSgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

hipsolverStatus_t cusolvergetrf(hipsolverHandle_t handle,
           int m,
           int n,
           double *A,
           int lda,
           double *Workspace,
           int *devIpiv,
           int *devInfo) {
    return hipsolverDnDgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

hipsolverStatus_t cusolvergetrf(hipsolverHandle_t handle,
           int m,
           int n,
           hipComplex *A,
           int lda,
           hipComplex *Workspace,
           int *devIpiv,
           int *devInfo) {
    return hipsolverDnCgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

hipsolverStatus_t cusolvergetrf(hipsolverHandle_t handle,
           int m,
           int n,
           hipDoubleComplex *A,
           int lda,
           hipDoubleComplex *Workspace,
           int *devIpiv,
           int *devInfo) {
    return hipsolverDnZgetrf(handle, m, n, A, lda, Workspace, devIpiv, devInfo);
}

hipsolverStatus_t
cusolvergetrs(hipsolverHandle_t handle,
           hipblasOperation_t trans,
           int n,
           int nrhs,
           const float *A,
           int lda,
           const int *devIpiv,
           float *B,
           int ldb,
           int *devInfo ) {
    return hipsolverDnSgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}

hipsolverStatus_t
cusolvergetrs(hipsolverHandle_t handle,
           hipblasOperation_t trans,
           int n,
           int nrhs,
           const double *A,
           int lda,
           const int *devIpiv,
           double *B,
           int ldb,
           int *devInfo ) {
    return hipsolverDnDgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}

hipsolverStatus_t
cusolvergetrs(hipsolverHandle_t handle,
           hipblasOperation_t trans,
           int n,
           int nrhs,
           const hipComplex *A,
           int lda,
           const int *devIpiv,
           hipComplex *B,
           int ldb,
           int *devInfo ) {
    return hipsolverDnCgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}

hipsolverStatus_t
cusolvergetrs(hipsolverHandle_t handle,
           hipblasOperation_t trans,
           int n,
           int nrhs,
           const hipDoubleComplex *A,
           int lda,
           const int *devIpiv,
           hipDoubleComplex *B,
           int ldb,
           int *devInfo ) {
    return hipsolverDnZgetrs(handle, trans, n, nrhs, A, lda, devIpiv, B, ldb, devInfo);
}

template<typename T>
void vat::memLUDecomposition(T* rows, T* b, T* x, const int dim) {
	hipsolverHandle_t cusolverH = NULL;
    hipStream_t stream = NULL;

    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = dim;
    const int lda = dim;
    const int ldb = dim;

    T LU[lda*m]; /* L and U */

    int Ipiv[m];      /* host copy of pivoting sequence */
    int info = 0;     /* host copy of error info */

    T *d_A = NULL; /* device copy of A */
    T *d_B = NULL; /* device copy of B */
    int *d_Ipiv = NULL; /* pivoting sequence */
    int *d_info = NULL; /* error info */
    int  lwork = 0;     /* size of workspace */
    T *d_work = NULL; /* device workspace for getrf */

    const int pivot_on = 1;

    printf("example of getrf \n");

    if (pivot_on){
        printf("pivot is on : compute P*A = L*U \n");
    }else{
        printf("pivot is off: compute A = L*U (not numerically stable)\n");
    }

    printf("A = (matlab base-1)\n");
    //printMatrix(m, m, A, lda, "A");
    printf("=====\n");

    printf("B = (matlab base-1)\n");
   // printMatrix(m, 1, B, ldb, "B");
    printf("=====\n");

/* step 1: create cusolver handle, bind a stream */
    status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);
    assert(hipSuccess == cudaStat1);

    status = hipsolverSetStream(cusolverH, stream);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

/* step 2: copy A to device */
    cudaStat1 = hipMalloc ((void**)&d_A, sizeof(T) * lda * m);
    cudaStat2 = hipMalloc ((void**)&d_B, sizeof(T) * m);
    cudaStat2 = hipMalloc ((void**)&d_Ipiv, sizeof(int) * m);
    cudaStat4 = hipMalloc ((void**)&d_info, sizeof(int));
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    cudaStat1 = hipMemcpy(d_A, rows, sizeof(T)*lda*m, hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, b, sizeof(T)*m, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);

    /* step 3: query working space of getrf */
    status = cusolverget_bufferSize(
        cusolverH,
        m,
        m,
        d_A,
        lda,
        &lwork);
    assert(HIPSOLVER_STATUS_SUCCESS == status);

    cudaStat1 = hipMalloc((void**)&d_work, sizeof(T)*lwork);
    assert(hipSuccess == cudaStat1);

/* step 4: LU factorization */
    if (pivot_on){
        status = cusolvergetrf(
            cusolverH,
            m,
            m,
            d_A,
            lda,
            d_work,
            d_Ipiv,
            d_info);
    }else{
        status = cusolvergetrf(
            cusolverH,
            m,
            m,
            d_A,
            lda,
            d_work,
            NULL,
            d_info);
    }
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    if (pivot_on){
    cudaStat1 = hipMemcpy(Ipiv , d_Ipiv, sizeof(int)*m, hipMemcpyDeviceToHost);
    }
    cudaStat2 = hipMemcpy(LU   , d_A   , sizeof(T)*lda*m, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);

    if ( 0 > info ){
        printf("%d-th parameter is wrong \n", -info);
        exit(1);
    }
    if (pivot_on){
        printf("pivoting sequence, matlab base-1\n");
        for(int j = 0 ; j < m ; j++){
            printf("Ipiv(%d) = %d\n", j+1, Ipiv[j]);
        }
    }
    //printf("L and U = (matlab base-1)\n");
    //printMatrix(m, m, LU, lda, "LU");
    //printf("=====\n");

    /*
 * step 5: solve A*X = B
 *       | 1 |       | -0.3333 |
 *   B = | 2 |,  X = |  0.6667 |
 *       | 3 |       |  0      |
 *
 */
    if (pivot_on){
        status = cusolvergetrs(
            cusolverH,
            HIPBLAS_OP_N,
            m,
            1, /* nrhs */
            d_A,
            lda,
            d_Ipiv,
            d_B,
            ldb,
            d_info);
    }else{
        status = cusolvergetrs(
            cusolverH,
            HIPBLAS_OP_N,
            m,
            1, /* nrhs */
            d_A,
            lda,
            NULL,
            d_B,
            ldb,
            d_info);
    }
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == status);
    assert(hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(x , d_B, sizeof(T)*m, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

    //printf("X = (matlab base-1)\n");
    //printMatrix(m, 1, X, ldb, "X");
    //printf("=====\n");

    if (d_A) hipFree(d_A);
    if (d_B) hipFree(d_B);
    if (d_Ipiv) hipFree(d_Ipiv);
    if (d_info) hipFree(d_info);
    if (d_work) hipFree(d_work);

    if (cusolverH) hipsolverDnDestroy(cusolverH);
    if (stream) hipStreamDestroy(stream);

    hipDeviceReset();
}
template void vat::memLUDecomposition<float>(float* rows, float* b, float* x, const int dim);
template void vat::memLUDecomposition<double>(double* rows, double* b, double* x, const int dim);
template void vat::memLUDecomposition<hipComplex>(hipComplex* rows, hipComplex* b, hipComplex* x, const int dim);
template void vat::memLUDecomposition<hipDoubleComplex>(hipDoubleComplex* rows, hipDoubleComplex* b, hipDoubleComplex* x, const int dim);
